
#include <hip/hip_runtime.h>
/* RuleTree:
rt :=
TFCall_tag( TFCall(TRC(TTensorI(MDDFT([ 4, 4, 4 ], 63, false), 4, APar, APar)), rec(
  fname := "batch3ddft4x4_cu_cu",
  params := [  ] )).withTags([ ASIMTKernelFlag(ASIMTGridDimY), ASIMTGridDimX, ASIMTBlockDimZ ]),
  TRC_SIMT( TRC(TTensorI(MDDFT([ 4, 4, 4 ], 63, false), 4, APar, APar)).withTags([ ASIMTKernelFlag(ASIMTGridDimY), ASIMTGridDimX, ASIMTBlockDimZ ]),
    IxA_SIMT( TTensorI(MDDFT([ 4, 4, 4 ], 63, false), 4, APar, APar).withTags([ ASIMTKernelFlag(ASIMTGridDimY), ASIMTGridDimX, ASIMTBlockDimZ ]),
      MDDFT_RowCol_3D_SIMT( MDDFT([ 4, 4, 4 ], 63, false).withTags([ ASIMTGridDimX, ASIMTBlockDimZ ]),
        MDDFT_tSPL_RowCol( MDDFT([ 4, 4 ], 15, false).withTags([ ASIMTBlockDimZ ]),
          IxB_AxI( TTensor(MDDFT([ 4 ], 3, false), MDDFT([ 4 ], 3, false)).withTags([ ASIMTBlockDimZ ]),
            TCompose_tag( TCompose([ TTensorI(MDDFT([ 4 ], 3, false), 4, APar, APar), TTensorI(MDDFT([ 4 ], 3, false), 4, AVec, AVec) ]).withTags([ ASIMTBlockDimZ ]),
              IxA_SIMT( TTensorI(MDDFT([ 4 ], 3, false), 4, APar, APar).withTags([ ASIMTBlockDimZ ]),
                MDDFT_Base( MDDFT([ 4 ], 3, false),
                  DFT_CT( DFT(4, 3),
                    DFT_Base( DFT(2, 1) ),
                    DFT_Base( DFT(2, 1) ) ) ) ),
              AxI_SIMT( TTensorI(MDDFT([ 4 ], 3, false), 4, AVec, AVec).withTags([ ASIMTBlockDimZ ]),
                MDDFT_Base( MDDFT([ 4 ], 3, false),
                  DFT_CT( DFT(4, 3),
                    DFT_Base( DFT(2, 1) ),
                    DFT_Base( DFT(2, 1) ) ) ) ) ) ) ),
        DFT_CT( DFT(4, 3),
          DFT_Base( DFT(2, 1) ),
          DFT_Base( DFT(2, 1) ) ) ) ) ) )
;
*/


/*
 * This code was generated by Spiral 8.3.0, www.spiral.net
 */

/*
__device__ double P1[512];
__device__ double P2[512];
*/

extern "C" __global__ void ker_batch3ddft4x4_cu_cu0(double  *X, double * P1) {
    double s25, s26, s27, s28, s29, s30, s31, s32, 
            t89, t90, t91, t92, t93, t94, t95, t96;
    int a84, a85, a86, a87, a88, a89, a90, a91;
    a84 = ((128*blockIdx.y) + (32*blockIdx.x) + (8*blockIdx.z));
    s25 = X[a84];
    a85 = (a84 + 1);
    s26 = X[a85];
    a86 = (a84 + 4);
    s27 = X[a86];
    a87 = (a84 + 5);
    s28 = X[a87];
    t89 = (s25 + s27);
    t90 = (s26 + s28);
    t91 = (s25 - s27);
    t92 = (s26 - s28);
    a88 = (a84 + 2);
    s29 = X[a88];
    a89 = (a84 + 3);
    s30 = X[a89];
    a90 = (a84 + 6);
    s31 = X[a90];
    a91 = (a84 + 7);
    s32 = X[a91];
    t93 = (s29 + s31);
    t94 = (s30 + s32);
    t95 = (s29 - s31);
    t96 = (s30 - s32);
    P1[a84] = (t89 + t93);
    P1[a85] = (t90 + t94);
    P1[a86] = (t89 - t93);
    P1[a87] = (t90 - t94);
    P1[a88] = (t91 + t96);
    P1[a89] = (t92 - t95);
    P1[a90] = (t91 - t96);
    P1[a91] = (t92 + t95);
}

extern "C" __global__ void ker_batch3ddft4x4_cu_cu1(double *P1, double *P2) {
    double s57, s58, s59, s60, s61, s62, s63, s64, 
            t137, t138, t139, t140, t141, t142, t143, t144;
    int a180, a181, a182;
    a180 = (128*blockIdx.y);
    a181 = (a180 + (8*blockIdx.z) + (2*blockIdx.x));
    s57 = P1[a181];
    s58 = P1[(a181 + 1)];
    s59 = P1[(a181 + 64)];
    s60 = P1[(a181 + 65)];
    t137 = (s57 + s59);
    t138 = (s58 + s60);
    t139 = (s57 - s59);
    t140 = (s58 - s60);
    s61 = P1[(a181 + 32)];
    s62 = P1[(a181 + 33)];
    s63 = P1[(a181 + 96)];
    s64 = P1[(a181 + 97)];
    t141 = (s61 + s63);
    t142 = (s62 + s64);
    t143 = (s61 - s63);
    t144 = (s62 - s64);
    a182 = (a180 + (2*blockIdx.z) + (32*blockIdx.x));
    P2[a182] = (t137 + t141);
    P2[(a182 + 1)] = (t138 + t142);
    P2[(a182 + 16)] = (t137 - t141);
    P2[(a182 + 17)] = (t138 - t142);
    P2[(a182 + 8)] = (t139 + t144);
    P2[(a182 + 9)] = (t140 - t143);
    P2[(a182 + 24)] = (t139 - t144);
    P2[(a182 + 25)] = (t140 + t143);
}

extern "C" __global__ void ker_batch3ddft4x4_cu_cu2(double  *Y, double *P2) {
    double s89, s90, s91, s92, s93, s94, s95, s96, 
            t185, t186, t187, t188, t189, t190, t191, t192;
    int a271, a272, a273;
    a271 = (128*blockIdx.y);
    a272 = (a271 + (8*blockIdx.z) + (32*blockIdx.x));
    s89 = P2[a272];
    s90 = P2[(a272 + 1)];
    s91 = P2[(a272 + 4)];
    s92 = P2[(a272 + 5)];
    t185 = (s89 + s91);
    t186 = (s90 + s92);
    t187 = (s89 - s91);
    t188 = (s90 - s92);
    s93 = P2[(a272 + 2)];
    s94 = P2[(a272 + 3)];
    s95 = P2[(a272 + 6)];
    s96 = P2[(a272 + 7)];
    t189 = (s93 + s95);
    t190 = (s94 + s96);
    t191 = (s93 - s95);
    t192 = (s94 - s96);
    a273 = (a271 + (32*blockIdx.z) + (2*blockIdx.x));
    Y[a273] = (t185 + t189);
    Y[(a273 + 1)] = (t186 + t190);
    Y[(a273 + 16)] = (t185 - t189);
    Y[(a273 + 17)] = (t186 - t190);
    Y[(a273 + 8)] = (t187 + t192);
    Y[(a273 + 9)] = (t188 - t191);
    Y[(a273 + 24)] = (t187 - t192);
    Y[(a273 + 25)] = (t188 + t191);
}

